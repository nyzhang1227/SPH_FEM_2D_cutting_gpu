// Copyright ETH Zurich, IWF
// SPH-FEM cutting simulations on the basis of mfree_iwf-ul_cut_gpu: 
// https://github.com/iwf-inspire/mfree_iwf-ul_cut_gpu
// Date: 2024.03.10

// You should have received a copy of the GNU General Public License
// along with mfree_iwf-ul_cut_gpu.  If not, see <http://www.gnu.org/licenses/>.

#include "particle_gpu.h"

particle_gpu::particle_gpu(unsigned int N) {
	hipMalloc((void **) &pos, sizeof(float2_t)*N);
	hipMalloc((void **) &vel, sizeof(float2_t)*N);
#ifdef TVF
	hipMalloc((void **) &vel_adv, sizeof(float2_t)*N);
#endif

	hipMalloc((void **) &h,   sizeof(float_t)*N);
	hipMalloc((void **) &rho, sizeof(float_t)*N);
	hipMalloc((void **) &p,   sizeof(float_t)*N);

	hipMalloc((void **) &S,   sizeof(float4_t)*N);
	hipMalloc((void **) &R,   sizeof(float4_t)*N);
	hipMalloc((void **) &fc,  sizeof(float2_t)*N);
	hipMalloc((void **) &ft,  sizeof(float2_t)*N);
	hipMalloc((void **) &n,   sizeof(float2_t)*N);

	hipMalloc((void**) &fixed, sizeof(float_t)*N);
	hipMalloc((void**) &blanked, sizeof(float_t)*N);
	hipMalloc((void**) &tool_particle, sizeof(float_t)*N);

	hipMalloc((void**) &eps_pl, sizeof(float_t)*N);
	hipMalloc((void**) &eps_pl_dot, sizeof(float_t)*N);
	hipMalloc((void**) &T, sizeof(float_t)*N);

	hipMalloc((void **) &pos_t, sizeof(float2_t)*N);
	hipMalloc((void **) &vel_t, sizeof(float2_t)*N);
#ifdef TVF
	hipMalloc((void **) &vel_adv_t, sizeof(float2_t)*N);
#endif
	hipMalloc((void **) &rho_t, sizeof(float_t)*N);
	hipMalloc((void **) &S_t,   sizeof(float4_t)*N);
	hipMalloc((void **) &T_t,   sizeof(float_t)*N);

	hipMalloc((void **) &v_der, sizeof(float4_t)*N);
	hipMalloc((void **) &S_der, sizeof(float4_t)*N);

#ifdef USE_FEM_TOOL
	hipMalloc((void**) &on_seg, sizeof(int) * N);
#endif 

	hipMalloc((void **) &idx, sizeof(int)*N);
	hipMalloc((void **) &hash, sizeof(int)*N);


	thrust::device_ptr<int> t_idx(this->idx);
	thrust::sequence(t_idx, t_idx+N);

	hipMemset(pos, 0, sizeof(float2_t)*N);
	hipMemset(vel,0, sizeof(float2_t)*N);

	hipMemset(h,0,   sizeof(float_t)*N);
	hipMemset(rho,0, sizeof(float_t)*N);
	hipMemset(p,0,   sizeof(float_t)*N);

	hipMemset(S,0,   sizeof(float4_t)*N);
	hipMemset(R,0,   sizeof(float4_t)*N);
	hipMemset(fc,0,  sizeof(float2_t)*N);
	hipMemset(ft,0,  sizeof(float2_t)*N);
	hipMemset(n,0,   sizeof(float2_t)*N);

	hipMemset(fixed,0,  sizeof(float_t)*N);
	hipMemset(blanked,0,  sizeof(float_t)*N);
	hipMemset(fixed,0,  sizeof(float_t)*N);
	hipMemset(tool_particle, 0, sizeof(float_t)*N);

	hipMemset(eps_pl, 0, sizeof(float_t)*N);
	hipMemset(eps_pl_dot, 0, sizeof(float_t)*N);
	hipMemset(T, 0, sizeof(float_t)*N);

	hipMemset(pos_t,0, sizeof(float2_t)*N);
	hipMemset(vel_t,0, sizeof(float2_t)*N);
	hipMemset(rho_t,0, sizeof(float_t)*N);
	hipMemset(S_t,0,   sizeof(float4_t)*N);
	hipMemset(T_t,0,   sizeof(float_t)*N);

	hipMemset(v_der, 0, sizeof(float4_t)*N);
	hipMemset(S_der, 0, sizeof(float4_t)*N);

#ifdef USE_FEM_TOOL
	/*
	int* on_seg_h = new int[N];
	for (int i = 0; i < N; i++) {
		on_seg_h[i] = -1;
	}
	hipMemcpy(on_seg, on_seg_h, sizeof(int) * N, hipMemcpyHostToDevice);
	*/
	hipMemset(on_seg, 0, sizeof(int) * N);
#endif 

	this->N = N;


}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *h, unsigned int N) : particle_gpu(N) {
	hipMemcpy(this->pos, pos,      sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, vel_init, sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->rho, rho,      sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->h,   h,        sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *h, float_t *fixed, unsigned int N)
: particle_gpu(pos, vel_init, rho, h, N) {
	hipMemcpy(this->fixed, fixed, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *T_init, float_t *h, float_t *fixed, unsigned int N)
: particle_gpu(pos, vel_init, rho, h, fixed, N) {
	hipMemcpy(this->T, T_init, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *T_init, float_t *h, float_t *fixed, float_t * tool_p, unsigned int N)
: particle_gpu(pos, vel_init, rho, T_init, h, fixed, N) {
	hipMemcpy(this->tool_particle, tool_p, sizeof(float_t)*N, hipMemcpyHostToDevice);
}

particle_gpu::particle_gpu(float2_t *pos, float2_t *vel_init, float_t *rho, float_t *h, float4_t *S, unsigned int N) : particle_gpu(N) {
	hipMemcpy(this->pos, pos,      sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->vel, vel_init, sizeof(float2_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->rho, rho,      sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->h,   h,        sizeof(float_t)*N, hipMemcpyHostToDevice);
	hipMemcpy(this->S,   S,        sizeof(float4_t)*N, hipMemcpyHostToDevice);
}
