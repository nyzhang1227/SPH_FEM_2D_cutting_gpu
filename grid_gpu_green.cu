#include "hip/hip_runtime.h"
// Copyright ETH Zurich, IWF
// SPH-FEM cutting simulations on the basis of mfree_iwf-ul_cut_gpu: 
// https://github.com/iwf-inspire/mfree_iwf-ul_cut_gpu
// Date: 2024.03.10

// You should have received a copy of the GNU General Public License
// along with mfree_iwf-ul_cut_gpu.  If not, see <http://www.gnu.org/licenses/>.

#include "grid_gpu_green.h"

//NOTE: its not worth it to bind particle arrays to textures, no runtime improvement measurable
//      	=> keep this version, code is more readable & compact
__global__ static void reorder_data_and_find_cell_start(
                                  const particle_gpu particles,
                                  grid_gpu_green::device_buffer buffer,
                                  int   *__restrict__ cell_start,
                                  int   *__restrict__ cell_end,
                                  int    num_cell) {
    extern __shared__ int shared_hash[];    // blockSize + 1 elements
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int hash = 0;

    // handle case when no. of particles not multiple of block size
    if (idx < num_cell) {
        hash = particles.hash[idx];

        // Load hash data into shared memory so that we can look
        // at neighboring particle's hash value without loading
        // two hash values per thread
        shared_hash[threadIdx.x+1] = hash;

        if (idx > 0 && threadIdx.x == 0) {
            // first thread in block must load neighbor particle hash
            shared_hash[0] = particles.hash[idx-1];
        }
    }

    __syncthreads();

    if (idx < num_cell) {
        if (idx == 0 || hash != shared_hash[threadIdx.x]) {
            cell_start[hash] = idx;
            if (idx > 0) {
                cell_end[shared_hash[threadIdx.x]] = idx;
            }
        }

        if (idx == num_cell - 1) {
            cell_end[hash] = idx + 1;
        }

        int sorted_index = particles.idx[idx];

        float2_t pos        = particles.pos[sorted_index];
        float2_t vel        = particles.vel[sorted_index];
#ifdef TVF
        float2_t vel_adv    = particles.vel_adv[sorted_index];
#endif
        float2_t fc         = particles.fc[sorted_index];
        float2_t ft         = particles.ft[sorted_index];
        float_t  h          = particles.h[sorted_index];
        float2_t n          = particles.n[sorted_index];
        float_t  rho        = particles.rho[sorted_index];
        float4_t S          = particles.S[sorted_index];
        float_t  eps_pl     = particles.eps_pl[sorted_index];
        float_t  eps_pl_dot = particles.eps_pl_dot[sorted_index];
        float_t  T          = particles.T[sorted_index];
        float_t  fixed      = particles.fixed[sorted_index];
        float_t  blanked    = particles.blanked[sorted_index];
        float_t  tool_particle = particles.tool_particle[sorted_index];

        float2_t pos_t  = particles.pos_t[sorted_index];
        float2_t vel_t  = particles.vel_t[sorted_index];
#ifdef TVF
        float2_t vel_adv_t  = particles.vel_adv_t[sorted_index];
#endif
        float_t  rho_t  = particles.rho_t[sorted_index];
        float4_t S_t    = particles.S_t[sorted_index];
        float_t  T_t    = particles.T_t[sorted_index];
#ifdef USE_FEM_TOOL
        int on_seg = particles.on_seg[sorted_index];
#endif

        buffer.pos[idx]        = pos;
        buffer.vel[idx]        = vel;
#ifdef TVF
        buffer.vel_adv[idx]        = vel_adv;
#endif
        buffer.fc[idx]         = fc;
        buffer.ft[idx]         = ft;
        buffer.h[idx]          = h;
        buffer.n[idx]          = n;
        buffer.rho[idx]        = rho;
        buffer.S[idx]          = S;
        buffer.eps_pl[idx]     = eps_pl;
        buffer.eps_pl_dot[idx] = eps_pl_dot;
        buffer.T[idx]          = T;
        buffer.fixed[idx]      = fixed;
        buffer.blanked[idx]    = blanked;
        buffer.tool_particle[idx]    = tool_particle;

        buffer.pos_t[idx]  = pos_t;
        buffer.vel_t[idx]  = vel_t;
#ifdef TVF
        buffer.vel_adv_t[idx]  = vel_adv_t;
#endif
        buffer.rho_t[idx]  = rho_t;
        buffer.S_t[idx]    = S_t;
        buffer.T_t[idx]    = T_t;
#ifdef USE_FEM_TOOL
        buffer.on_seg[idx] = on_seg;
#endif
    }
}

__global__ static void copy_from_buffer(particle_gpu particles, grid_gpu_green::device_buffer buffer, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= N) return;

    particles.pos[idx]        = buffer.pos[idx];
    particles.vel[idx]        = buffer.vel[idx];
#ifdef TVF
    particles.vel_adv[idx]        = buffer.vel_adv[idx];
#endif
    particles.fc[idx]         = buffer.fc[idx];
    particles.ft[idx]         = buffer.ft[idx];
    particles.h[idx]          = buffer.h[idx];
    particles.n[idx]          = buffer.n[idx];
    particles.rho[idx]        = buffer.rho[idx];
    particles.S[idx]          = buffer.S[idx];
    particles.eps_pl[idx]     = buffer.eps_pl[idx];
    particles.eps_pl_dot[idx] = buffer.eps_pl_dot[idx];
    particles.T[idx]          = buffer.T[idx];
    particles.fixed[idx]      = buffer.fixed[idx];
    particles.blanked[idx]    = buffer.blanked[idx];
    particles.tool_particle[idx]    = buffer.tool_particle[idx];

    particles.pos_t[idx]  = buffer.pos_t[idx];
    particles.vel_t[idx]  = buffer.vel_t[idx];
#ifdef TVF
    particles.vel_adv_t[idx]  = buffer.vel_adv_t[idx];
#endif
    particles.rho_t[idx]  = buffer.rho_t[idx];
    particles.S_t[idx]    = buffer.S_t[idx];
    particles.T_t[idx]    = buffer.T_t[idx];
#ifdef USE_FEM_TOOL
    particles.on_seg[idx] = buffer.on_seg[idx];
#endif
}

static void double_buffer(particle_gpu *particles, grid_gpu_green::device_buffer *buffer) {
	std::swap(particles->pos,buffer->pos);
    std::swap(particles->vel, buffer->vel);
#ifdef TVF
    std::swap(particles->vel_adv, buffer->vel_adv);
#endif
    std::swap(particles->fc, buffer->fc);
    std::swap(particles->ft, buffer->ft);
    std::swap(particles->h, buffer->h);
    std::swap(particles->n, buffer->n);
    std::swap(particles->rho, buffer->rho);
    std::swap(particles->S, buffer->S);
    std::swap(particles->eps_pl, buffer->eps_pl);
    std::swap(particles->eps_pl_dot, buffer->eps_pl_dot);
    std::swap(particles->T, buffer->T);
    std::swap(particles->fixed, buffer->fixed);
    std::swap(particles->blanked, buffer->blanked);
    std::swap(particles->tool_particle, buffer->tool_particle);

    std::swap(particles->pos_t, buffer->pos_t);
    std::swap(particles->vel_t, buffer->vel_t);
#ifdef TVF
    std::swap(particles->vel_adv_t, buffer->vel_adv_t);
#endif
    std::swap(particles->rho_t, buffer->rho_t);
    std::swap(particles->S_t, buffer->S_t);
    std::swap(particles->T_t, buffer->T_t);
#ifdef USE_FEM_TOOL
    std::swap(particles->on_seg, buffer->on_seg);
#endif
}

template<class T>
static void do_sort(T* particles, grid_gpu_green::device_buffer *buffer, bool use_double_buffer, int *cell_start, int *cell_end, int max_cell) {
	//index vec

	thrust::device_ptr<int> t_idx(particles->idx);
	thrust::sequence(t_idx, t_idx+particles->N);

	//keys to sort by
	thrust::device_ptr<int> t_hashes(particles->hash);
	thrust::sort_by_key(t_hashes, t_hashes+particles->N, t_idx);

	hipMemset(cell_start, 0xffffffff, max_cell*sizeof(int));

	const unsigned int block_size = 512;
	dim3 dG((particles->N + block_size-1) / block_size);
	dim3 dB(block_size);
	unsigned int shared_mem_size = (block_size+1)*sizeof(int);

	reorder_data_and_find_cell_start<<< dG, dB, shared_mem_size>>>(*particles, *buffer, cell_start, cell_end, particles->N);

	if (use_double_buffer) {
		double_buffer(particles, buffer);
	} else {
		copy_from_buffer<<<dG, dB>>>(*particles, *buffer, particles->N);
	}
}

void grid_gpu_green::sort(particle_gpu *particles, tool *tool) const {
	do_sort(particles, m_buffer, m_buffer_method == buffer_method::swap, m_cell_start, m_cell_end, m_num_cell);
}

void grid_gpu_green::get_cells(particle_gpu *particles, int *cell_start, int *cell_end)  {
	hipMemcpy(cell_start, m_cell_start, sizeof(int)*m_num_cell, hipMemcpyDeviceToDevice);
	hipMemcpy(cell_end,   m_cell_end,   sizeof(int)*m_num_cell, hipMemcpyDeviceToDevice);
}

void grid_gpu_green::alloc_buffer(int num_cell, int num_part) {
	m_buffer = new grid_gpu_green::device_buffer();

	hipMalloc((void**) &m_buffer->pos, sizeof(float2_t)*num_part);
	hipMalloc((void**) &m_buffer->vel, sizeof(float2_t)*num_part);
#ifdef TVF
	hipMalloc((void**) &m_buffer->vel_adv, sizeof(float2_t)*num_part);
#endif
	hipMalloc((void**) &m_buffer->fc, sizeof(float2_t)*num_part);
	hipMalloc((void**) &m_buffer->ft, sizeof(float2_t)*num_part);
	hipMalloc((void**) &m_buffer->h, sizeof(float_t)*num_part);
    hipMalloc((void**) &m_buffer->n, sizeof(float2_t)*num_part);
	hipMalloc((void**) &m_buffer->rho, sizeof(float_t)*num_part);
	hipMalloc((void**) &m_buffer->S, sizeof(float4_t)*num_part);
	hipMalloc((void**) &m_buffer->eps_pl, sizeof(float_t)*num_part);
	hipMalloc((void**) &m_buffer->eps_pl_dot, sizeof(float_t)*num_part);
	hipMalloc((void**) &m_buffer->T, sizeof(float_t)*num_part);
	hipMalloc((void**) &m_buffer->fixed, sizeof(float_t)*num_part);
	hipMalloc((void**) &m_buffer->blanked, sizeof(float_t)*num_part);
	hipMalloc((void**) &m_buffer->tool_particle, sizeof(float_t)*num_part);

	hipMalloc((void**) &m_buffer->pos_t, sizeof(float2_t)*num_part);
	hipMalloc((void**) &m_buffer->vel_t, sizeof(float2_t)*num_part);
#ifdef TVF
	hipMalloc((void**) &m_buffer->vel_adv_t, sizeof(float2_t)*num_part);
#endif
	hipMalloc((void**) &m_buffer->rho_t, sizeof(float_t)*num_part);
	hipMalloc((void**) &m_buffer->S_t, sizeof(float4_t)*num_part);
	hipMalloc((void**) &m_buffer->T_t, sizeof(float_t)*num_part);
#ifdef USE_FEM_TOOL
    hipMalloc((void**) &m_buffer->on_seg, sizeof(int) * num_part);
#endif

	hipMalloc((void**) &m_cell_start, sizeof(int)*num_cell);
	hipMalloc((void**) &m_cell_end,   sizeof(int)*num_cell);

}

grid_gpu_green::grid_gpu_green(unsigned int max_cell, unsigned int N) :
	grid_base(max_cell, N) {

	alloc_buffer(max_cell, N);
}

grid_gpu_green::grid_gpu_green(int num_part, float2_t bbmin, float2_t bbmax, float_t h) :
	grid_base(num_part, bbmin, bbmax, h) {

	alloc_buffer(m_max_cell, num_part);
}
